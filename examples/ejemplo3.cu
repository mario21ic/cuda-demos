// Hilos y Bloques
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define BLOCK 10
#define N 25

// Kernel Multibloque
__global__
void suma(int *vector_1, int *vector_2, int*vector_suma, int n)
{
    // identificador de hilo
    int myID = threadIdx.x + blockDim.x * blockIdx.x;

    // solo para n hilos
    if (myID < n) {
        // inicializamos el vector 2
        vector_2[myID] = (n -1) - myID;
        // escritura de resultados
        vector_suma[myID] = vector_1[myID] + vector_2[myID];
    }
}

int main(int argc, char** argv) {
    // declaraciones
    int *hst_vector1, *hst_vector2, *hst_resultado;
    int *dev_vector1, *dev_vector2, *dev_resultado;

    // reserva en el host
    hst_vector1 = (int*)malloc(N * sizeof(int));
    hst_vector2 = (int*)malloc(N * sizeof(int));
    hst_resultado = (int*)malloc(N * sizeof(int));

    // reserva en el device
    hipMalloc((void**)&dev_vector1, N * sizeof(int));
    hipMalloc((void**)&dev_vector2, N * sizeof(int));
    hipMalloc((void**)&dev_resultado, N * sizeof(int));

    // inicializacion de vectores
    for (int i=0; i<N; i++) {
        hst_vector1[i] = i;
        hst_vector2[i] = 0;
    }

    // copia de datos hacia el device
    hipMemcpy(dev_vector1, hst_vector1, N*sizeof(int), hipMemcpyHostToDevice);

    // calculamos nro de bloques
    int bloques = N / BLOCK;
    /* Si el tamano del vector no es multiplo del 
    tamano del bloque, lanzamos otro bloque
    (tendremos hilos sobrantes) */
    if ((N % BLOCK) != 0) {
        bloques = bloques + 1;
    }

    printf("> Vector de %d elementos\n", N);
    printf("> Lanzamiento con %d bloques de %d hilos (%d hilos)\n", bloques, BLOCK, bloques*BLOCK);

    // Lanzamiento del Kernel
    suma <<< bloques, BLOCK >>> (dev_vector1, dev_vector2, dev_resultado, N);

    // recogida de datos desde el device
    hipMemcpy(hst_vector2, dev_vector2, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hst_resultado, dev_resultado, N*sizeof(int), hipMemcpyDeviceToHost);

    // impresion de resultados
    printf("VECTOR 1:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_vector1[i]);
    }
    printf("\n");

    printf("VECTOR 2:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_vector2[i]);
    }
    printf("\n");

    printf("SUMA:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_resultado[i]);
    }
    printf("\n");

    printf("****");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}
