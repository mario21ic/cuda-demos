#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 8

__global__
void suma(int *vector_1, int *vector_2, int*vector_suma, int n)
{
    // identificador de hilo
    int myID = threadIdx.x;
    // inicializamos el vector 2
    vector_2[myID] = (n -1) - myID;
    // escritura de resultados
    vector_suma[myID] = vector_1[myID] + vector_2[myID];
}

int main(int argc, char** argv) {
    // declaraciones
    int *hst_vector1, *hst_vector2, *hst_resultado;
    int *dev_vector1, *dev_vector2, *dev_resultado;

    // reserva en el host
    hst_vector1 = (int*)malloc(N * sizeof(int));
    hst_vector2 = (int*)malloc(N * sizeof(int));
    hst_resultado = (int*)malloc(N * sizeof(int));

    // reserva en el device
    hipMalloc((void**)&dev_vector1, N * sizeof(int));
    hipMalloc((void**)&dev_vector2, N * sizeof(int));
    hipMalloc((void**)&dev_resultado, N * sizeof(int));

    // inicializacion de vectores
    for (int i=0; i<N; i++) {
        hst_vector1[i] = i;
        hst_vector2[i] = 0;
    }

    // copia de datos hacia el device
    hipMemcpy(dev_vector1, hst_vector1, N*sizeof(int), hipMemcpyHostToDevice);

    // lanzamiento del kernel
    suma <<< 1, N >>> (dev_vector1, dev_vector2, dev_resultado, N);

    // recogida de datos desde el device
    hipMemcpy(hst_vector2, dev_vector2, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hst_resultado, dev_resultado, N*sizeof(int), hipMemcpyDeviceToHost);

    // impresion de resultados
    printf("VECTOR 1:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_vector1[i]);
    }
    printf("\n");

    printf("VECTOR 2:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_vector2[i]);
    }
    printf("\n");

    printf("SUMA:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_resultado[i]);
    }
    printf("\n");

    printf("****");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}
