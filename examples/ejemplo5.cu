// Temporizador de un Kernel
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

# define N 24

// Funcion que suma de dos vectores de n elementos
__global__
void suma(int *vector_1, int *vector_2, int*vector_suma, int n)
{
    // identificador de hilo
    int myID = threadIdx.x;

    // obtenemos el vector invertido
    vector_2[myID] = vector_1[n -1 - myID];

    // escritura de resultados
    vector_suma[myID] = vector_1[myID] + vector_2[myID];
}


int main(int argc, char** argv) {
    // declaraciones
    int *hst_vector1, *hst_vector2, *hst_resultado;
    int *dev_vector1, *dev_vector2, *dev_resultado;

    // reserva en el host
    hst_vector1 = (int*)malloc(N * sizeof(int));
    hst_vector2 = (int*)malloc(N * sizeof(int));
    hst_resultado = (int*)malloc(N * sizeof(int));

    // reserva en el device
    hipMalloc((void**)&dev_vector1, N * sizeof(int));
    hipMalloc((void**)&dev_vector2, N * sizeof(int));
    hipMalloc((void**)&dev_resultado, N * sizeof(int));

    // inicializacion de vectores
    for (int i=0; i<N; i++) {
        hst_vector1[i] = rand() % 10;
        hst_vector2[i] = 0;
    }
    printf("> Vector de %d elementos\n", N);
    printf("> Lanzamiento con %d bloque de %d hilos\n", 1, N);

    // Temporizacion
    // Declaracion de eventos
    hipEvent_t start;
    hipEvent_t stop;

    // Creacion de eventos
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copia de datos hacia el device
    hipMemcpy(dev_vector1, hst_vector1, N*sizeof(int), hipMemcpyHostToDevice);

    // marca de inicio
    hipEventRecord(start, 0);

    // Lanzamos Kernel de un solo bloque y 24 hilos
    suma <<< 1, N >>> (dev_vector1, dev_vector2, dev_resultado, N);

    // marca de final
    hipEventRecord(stop, 0);

    // recogida de datos desde el device
    hipMemcpy(hst_vector2, dev_vector2, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(hst_resultado, dev_resultado, N*sizeof(int), hipMemcpyDeviceToHost);

    // sincronizacion GPU-CPU
    hipEventSynchronize(stop);

    // Calculo del tiempo en milisegundos
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\n> Tiempo de ejecucion: %f ms\n", elapsedTime);

    // impresion de resultados
    printf("VECTOR 1:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_vector1[i]);
    }
    printf("\n");

    printf("VECTOR 2:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_vector2[i]);
    }
    printf("\n");

    printf("SUMA:\n");
    for (int i=0; i<N; i++) {
        printf("%.2d ", hst_resultado[i]);
    }
    printf("\n");

    printf("****");
    printf("<pulsa [INTRO] para finalizar>");
    getchar();
    return 0;
}
