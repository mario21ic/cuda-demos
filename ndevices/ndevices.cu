#include <hip/hip_runtime.h>
#include <stdio.h> 

int main() {
  int driver_version = 0, runtime_version = 0;
  hipDriverGetVersion(&driver_version);
  hipRuntimeGetVersion(&runtime_version);

  printf("Driver Version: %d\n Runtime Version: %d\n", \
    driver_version, runtime_version);

  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  MultiProccessor: %d\n", prop.multiProcessorCount);
    printf("  Total GlobalMemory mb: %zu\n", prop.totalGlobalMem/(1024*1024));
    printf("  Total ConstMemory mb: %zu\n", prop.totalConstMem/(1024*1024));
    printf("  ---\n");

    printf("  sharedMemPerBlock: %zu\n", prop.sharedMemPerBlock);
    printf("  regsPerBlock: %d\n", prop.regsPerBlock);
    printf("  maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
    printf("  ---\n");

    printf("  sharedMemPerMultiprocessor: %zu\n", prop.sharedMemPerMultiprocessor);
    printf("  regsPerMultiprocessor: %d\n", prop.regsPerMultiprocessor);
    printf("  maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  ---\n");

    printf("  warpSize: %d\n", prop.warpSize);
    printf("  memPitch: %d\n", prop.memPitch);
    printf("  maxThreadsDim[3]: %d\n", prop.maxThreadsDim[3]);
    printf("  maxGridSize[3]: %d\n", prop.maxGridSize[3]);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
