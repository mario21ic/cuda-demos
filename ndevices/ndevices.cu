#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute Capability: %d.%d\n", prop.major, prop.minor);
    printf("  MultiProccessor: %d\n", prop.multiProcessorCount);
    printf("  maxThreadsPerMultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("  regsPerMultiprocessor: %d\n", prop.regsPerMultiprocessor);
    printf("  Total Memory mb: %zu\n", prop.totalGlobalMem/(1024*1024));
    printf("  sharedMemPerMultiprocessor: %zu\n", prop.sharedMemPerMultiprocessor);
    printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
